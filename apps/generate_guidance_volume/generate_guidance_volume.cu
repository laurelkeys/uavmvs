#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2016-2018, Nils Moehrle
 * All rights reserved.
 *
 * This software may be modified and distributed under the terms
 * of the BSD 3-Clause license. See the LICENSE.txt file for details.
 */

#include <cassert>
#include <iostream>

#if 0
#include "fmt/format.h"
#endif

#include "util/system.h"
#include "util/arguments.h"
#include "util/file_system.h"

#include "mve/camera.h"
#include "mve/mesh_io_ply.h"
#include "mve/image_io.h"
#include "mve/image_tools.h"

#include "acc/primitives.h"

#include "cacc/math.h"
#include "cacc/util.h"
#include "cacc/bvh_tree.h"
#include "cacc/nnsearch.h"
#include "cacc/point_cloud.h"

#include "util/io.h"
#include "util/cio.h"
#include "util/progress_counter.h"
#include "util/itos.h"

#include "geom/sphere.h"
#include "geom/volume_io.h"

#include "eval/kernels.h"

constexpr float lowest = std::numeric_limits<float>::lowest();

struct Arguments {
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string airspace_mesh;
    std::string ovolume;
    float resolution;
    float max_distance;
    float min_altitude;
    float max_altitude;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_minnum(4);
    args.set_nonopt_maxnum(4);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] PROXY_MESH PROXY_CLOUD AIRSPACE_MESH OUT_VOLUME");
    args.set_description("TODO");
    args.add_option('r', "resolution", true, "guidance volume resolution [1.0]");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [80.0]");
    args.add_option('\0', "min-altitude", true, "minimum altitude [0.0]");
    args.add_option('\0', "max-altitude", true, "maximum altitude [100.0]");
    args.parse(argc, argv);

    Arguments conf;
    conf.proxy_mesh = args.get_nth_nonopt(0);
    conf.proxy_cloud = args.get_nth_nonopt(1);
    conf.airspace_mesh = args.get_nth_nonopt(2);
    conf.ovolume = args.get_nth_nonopt(3);
    conf.resolution = 1.0f;
    conf.max_distance = 80.0f;
    conf.min_altitude = 0.0f;
    conf.max_altitude = 100.0f;

    for (util::ArgResult const* i = args.next_option();
         i != 0; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'r':
            conf.resolution = i->get_arg<float>();
        break;
        case '\0':
            if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else if (i->opt->lopt == "min-altitude") {
                conf.min_altitude = i->get_arg<float>();
            } else if (i->opt->lopt == "max-altitude") {
                conf.max_altitude = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char **argv) {
    util::system::register_segfault_handler();
    util::system::print_build_timestamp(argv[0]);

    Arguments args = parse_args(argc, argv);

    int device = cacc::select_cuda_device(3, 5);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }

    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(args.airspace_mesh);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: " << e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & verts = mesh->get_vertices();

    //TODO merge with proxy mesh generation code
    acc::AABB<math::Vec3f> aabb = acc::calculate_aabb(verts);

    assert(acc::valid(aabb) && acc::volume(aabb) > 0.0f);

    int width = (aabb.max[0] - aabb.min[0]) / args.resolution + 1.0f;
    int height = (aabb.max[1] - aabb.min[1]) / args.resolution + 1.0f;
    int depth = args.max_altitude / args.resolution + 1.0f;

    std::cout << width << "x" << height << "x" << depth << std::endl;

    /* Create height map. */
    mve::FloatImage::Ptr hmap = mve::FloatImage::create(width, height, 1);
    hmap->fill(lowest);
    for (std::size_t i = 0; i < verts.size(); ++i) {
        math::Vec3f vertex = verts[i];
        int x = (vertex[0] - aabb.min[0]) / args.resolution;
        assert(0 <= x && x < width);
        int y = (vertex[1] - aabb.min[1]) / args.resolution;
        assert(0 <= y && y < height);
        float height = vertex[2];
        float z = hmap->at(x, y, 0);
        if (z > height) continue;

        hmap->at(x, y, 0) = height;
    }

    /* Estimate ground level and normalize height map */
    float ground_level = std::numeric_limits<float>::max();
    #pragma omp parallel for reduction(min:ground_level)
    for (int i = 0; i < hmap->get_value_amount(); ++i) {
        float height = hmap->at(i);
        if (height != lowest && height < ground_level) {
            ground_level = height;
        }
    }

    #pragma omp parallel for
    for (int i = 0; i < hmap->get_value_amount(); ++i) {
        float height = hmap->at(i);
        hmap->at(i) = (height != lowest) ? height - ground_level : 0.0f;
    }
    //ODOT merge with proxy mesh generation code

    Volume<std::uint32_t>::Ptr volume;
    volume = Volume<std::uint32_t>::create(width, height, depth, aabb.min, aabb.max);
    std::vector<math::Vector<std::uint32_t, 3> > sample_positions;
    sample_positions.reserve(volume->num_positions());

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {

            float px = (x - args.resolution / 2.0f) * args.resolution + aabb.min[0];
            float py = (y - args.resolution / 2.0f) * args.resolution + aabb.min[1];

            float fz = std::max(hmap->at(x, y, 0), args.min_altitude);

            for (int z = 0; z < depth; ++z) {
                float pz = ground_level + z * args.resolution;

                if (pz < fz) continue;

                sample_positions.emplace_back(x, y, z);
            }
        }
    }

    uint num_verts;
    cacc::KDTree<3u, cacc::DEVICE>::Ptr dkd_tree;
    {
        mve::TriangleMesh::Ptr sphere = generate_sphere_mesh(1.0f, 3u);
        std::vector<math::Vec3f> const & verts = sphere->get_vertices();
        num_verts = verts.size();
        acc::KDTree<3u, uint>::Ptr kd_tree = acc::KDTree<3, uint>::create(verts);
        dkd_tree = cacc::KDTree<3u, cacc::DEVICE>::create<uint>(kd_tree);
    }

    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    {
        cacc::PointCloud<cacc::HOST>::Ptr cloud;
        cloud = load_point_cloud(args.proxy_cloud);
        dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);
    }

    mve::CameraInfo cam;
    cam.flen = 0.86f;
    math::Matrix3f calib;

    std::size_t num_samples = sample_positions.size() * 128ull * 45ull;

#if 0
    std::string task = fmt::format("Sampling 5D volume at {} positions", litos(num_samples));
#else
    std::string task("Sampling 5D volume at ");
    task += litos(num_samples);
    task += std::string(" positions");
#endif
    ProgressCounter counter(task, sample_positions.size());

    #pragma omp parallel
    {
        cacc::set_cuda_device(device);

        hipStream_t stream;
        hipStreamCreate(&stream);

        int width = 1920;
        int height = 1080;
        cam.fill_calibration(calib.begin(), width, height);

        cacc::Array<float, cacc::DEVICE>::Ptr dobs_hist;
        dobs_hist = cacc::Array<float, cacc::DEVICE>::create(num_verts, stream);

        cacc::Image<float, cacc::DEVICE>::Ptr dhist;
        dhist = cacc::Image<float, cacc::DEVICE>::create(128, 45, stream);
        cacc::Image<float, cacc::HOST>::Ptr hist;
        hist = cacc::Image<float, cacc::HOST>::create(128, 45, stream);

        #pragma omp for schedule(dynamic)
        for (std::size_t i = 0; i < sample_positions.size(); ++i) {
            counter.progress<ETA>();

            dobs_hist->null();
            {
                dim3 grid(cacc::divup(dcloud->cdata().num_vertices, KERNEL_BLOCK_SIZE));
                dim3 block(KERNEL_BLOCK_SIZE);
                populate_spherical_histogram<<<grid, block, 0, stream>>>(
                    cacc::Vec3f(volume->position(sample_positions[i]).begin()),
                    args.max_distance, dbvh_tree->accessor(), dcloud->cdata(),
                    dkd_tree->accessor(), dobs_hist->cdata());
            }

            {
                dim3 grid(cacc::divup(128, KERNEL_BLOCK_SIZE), 45);
                dim3 block(KERNEL_BLOCK_SIZE);
                evaluate_spherical_histogram<<<grid, block, 0, stream>>>(
                    cacc::Mat3f(calib.begin()), width, height,
                    dkd_tree->accessor(), dobs_hist->cdata(), dhist->cdata());
            }

            *hist = *dhist;
            cacc::Image<float, cacc::HOST>::Data data = hist->cdata();

            hist->sync();

            mve::FloatImage::Ptr image = mve::FloatImage::create(128, 45, 1);
            float const * begin = data.data_ptr;
            float const * end = data.data_ptr + data.width * data.height;
            std::copy(begin, end, image->get_data_pointer());
            volume->at(sample_positions[i]) = image;

            counter.inc();
        }
        hipStreamDestroy(stream);
    }

    save_volume<std::uint32_t>(volume, args.ovolume);

    return EXIT_SUCCESS;
}
